#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <string>
#include <fstream>
#include <chrono>
#include <cstring>
#include <algorithm>
#include <hip/hip_runtime.h>
#include <iomanip>
#include <stdio.h>

#ifdef _WIN32
#include <windows.h>
#endif

#include "crypto_kernels.cuh"

#define CUDA_CHECK(x) do { hipError_t e = x; if(e != hipSuccess) { \
    fprintf(stderr, "CUDA ERROR: %s (%s:%d)\n", hipGetErrorString(e), __FILE__, __LINE__); exit(1);}} while(0)

// =============================================================================
// KERNEL
// =============================================================================
__global__ void generation_and_validation_kernel(
    const char* passwords_data, const int* password_lengths, int password_maxlen,
    const uint8_t* salt, const uint8_t* ciphertext, size_t ciphertext_len,
    int sha_iterations, int kdf_iterations, int key_len_bytes, int iv_len_bytes,
    int* found_flag, char* found_password, int num_passwords,
    int debug_idx
) 
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num_passwords || (*found_flag && debug_idx == -1)) return;

    // --- PART 1: Key & IV Generation ---
    const char* pw = passwords_data + idx * password_maxlen;
    int pw_len = password_lengths[idx];
    char local_pw[128];
    for(int i=0; i<pw_len; ++i) local_pw[i] = pw[i];

    char sha512_hex_output[129] = {0};
    iterative_sha512(local_pw, pw_len, sha512_hex_output, sha_iterations);
    
    uint8_t kdf_key_iv[144];
    int kdf_key_iv_len = 0;
    uint8_t md5_prev[16] = {0};
    bool has_prev = false;
    int total_bytes_needed = key_len_bytes + iv_len_bytes;

    while (kdf_key_iv_len < total_bytes_needed) {
        uint8_t block[256];
        int block_len = 0;
        if (has_prev) for (int i = 0; i < 16; ++i) block[block_len++] = md5_prev[i];
        for (int i = 0; i < 128; ++i) block[block_len++] = (uint8_t)sha512_hex_output[i];
        for (int i = 0; i < 8; ++i) block[block_len++] = salt[i];

        md5_cuda(block, block_len, md5_prev);
        for (int i = 1; i < kdf_iterations; ++i) md5_cuda(md5_prev, 16, md5_prev);

        int copy_len = 16;
        if (kdf_key_iv_len + copy_len > total_bytes_needed) copy_len = total_bytes_needed - kdf_key_iv_len;
        for (int i = 0; i < copy_len; ++i) kdf_key_iv[kdf_key_iv_len + i] = md5_prev[i];
        
        kdf_key_iv_len += copy_len;
        has_prev = true;
    }

    uint8_t* generated_key = kdf_key_iv;
    uint8_t* generated_iv = kdf_key_iv + key_len_bytes;
    
    // --- PART 2: AES Decryption & Validation ---
    uint8_t round_key[240 * 4];
    uint8_t local_iv[AES_BLOCKLEN];
    uint8_t decrypted_buffer[4096]; 
    uint8_t aes_rounds = 38, aes_Nk = 32;

    KeyExpansion(round_key, generated_key, aes_rounds, aes_Nk);
    for(int i=0; i<AES_BLOCKLEN; ++i) local_iv[i] = generated_iv[i];
    for(size_t i=0; i<ciphertext_len; ++i) decrypted_buffer[i] = ciphertext[i];

    uint8_t storeNextIv[AES_BLOCKLEN];
    for (size_t i = 0; i < ciphertext_len; i += AES_BLOCKLEN) {
        for(int j=0; j<AES_BLOCKLEN; ++j) storeNextIv[j] = decrypted_buffer[i+j];
        InvCipher((state_t*)(decrypted_buffer + i), round_key, aes_rounds);
        XorWithIv(decrypted_buffer + i, local_iv);
        for(int j=0; j<AES_BLOCKLEN; ++j) local_iv[j] = storeNextIv[j];
    }
    
    const char* pattern = "\"kty\":\"RSA\"";
    for (size_t i = 0; i + 10 <= ciphertext_len; ++i) {
        bool match = true;
        for(int j=0; j<10; ++j) if (decrypted_buffer[i+j] != pattern[j]) { match = false; break; }
        if (match) {
            if (atomicExch(found_flag, 1) == 0) {
                for(int k=0; k < pw_len; ++k) found_password[k] = local_pw[k];
                found_password[pw_len] = '\0';
            }
            return;
        }
    }
}

// =============================================================================
// HOST CODE
// =============================================================================
bool load_and_decode_message(const std::string& filename, std::vector<uint8_t>& salt, std::vector<uint8_t>& ciphertext) {
    std::ifstream infile(filename);
    if (!infile) return false;
    std::string b64_data((std::istreambuf_iterator<char>(infile)), std::istreambuf_iterator<char>());
    infile.close();
    b64_data.erase(std::remove(b64_data.begin(), b64_data.end(), '\n'), b64_data.end());
    b64_data.erase(std::remove(b64_data.begin(), b64_data.end(), '\r'), b64_data.end());

    std::string chars = "ABCDEFGHIJKLMNOPQRSTUVWXYZabcdefghijklmnopqrstuvwxyz0123456789+/";
    std::vector<uint8_t> decoded;
    int val = 0, bits = 0;
    for (char c : b64_data) {
        if (c == '=') break;
        auto pos = chars.find(c);
        if (pos == std::string::npos) continue;
        val = (val << 6) | pos;
        bits += 6;
        if (bits >= 8) {
            bits -= 8;
            decoded.push_back(val >> bits);
        }
    }
    if (decoded.size() < 16 || std::string(decoded.begin(), decoded.begin() + 8) != "Salted__") return false;
    salt.assign(decoded.begin() + 8, decoded.begin() + 16);
    ciphertext.assign(decoded.begin() + 16, decoded.end());
    return true;
}

int main(int argc, char* argv[]) {
    // *** NEU: Überprüfe, ob ein Dateiname als Argument übergeben wurde ***
    if (argc < 2) {
        std::cerr << "Usage: " << argv[0] << " <input_filename.txt>\n";
        return 1;
    }
    std::string input_filename = argv[1];
    std::cout << "Loading passwords from: " << input_filename << "\n";

    std::vector<std::string> passwords;
    std::ifstream infile(input_filename);
    if (!infile) {
        std::cerr << "Error: Could not open password file '" << input_filename << "'.\n";
        return 1;
    }
    std::string line;
    while (std::getline(infile, line)) {
        while (!line.empty() && (line.back() == '\n' || line.back() == '\r')) line.pop_back();
        if (!line.empty()) passwords.push_back(line);
    }
    
    if (passwords.empty()) { std::cerr << "Error: No passwords found in '" << input_filename << "'.\n"; return 1; }
    std::cout << passwords.size() << " passwords loaded.\n\n";

    std::vector<uint8_t> salt, ciphertext;
    if (!load_and_decode_message("message.b64", salt, ciphertext)) {
        std::cerr << "Error: 'message.b64' could not be read or decoded.\n"; return 1;
    }
    std::cout << "Message 'message.b64' loaded.\n";
    std::cout << " - Salt (HEX): ";
    for(uint8_t b : salt) std::cout << std::hex << std::setw(2) << std::setfill('0') << (int)b;
    std::cout << "\n - Ciphertext length: " << ciphertext.size() << " Bytes\n\n";

    const int PW_MAXLEN = 128;
    std::vector<char> h_pw_data(passwords.size() * PW_MAXLEN, 0);
    std::vector<int>  h_pw_lengths(passwords.size());
    for(size_t i=0; i<passwords.size(); ++i) {
        int len = std::min((int)passwords[i].size(), PW_MAXLEN);
        memcpy(&h_pw_data[i * PW_MAXLEN], passwords[i].c_str(), len);
        h_pw_lengths[i] = len;
    }

    char* d_pw_data; int* d_pw_lengths; uint8_t* d_salt; uint8_t* d_ciphertext;
    int* d_found_flag; char* d_found_password;
    CUDA_CHECK(hipMalloc(&d_pw_data, h_pw_data.size()));
    CUDA_CHECK(hipMalloc(&d_pw_lengths, h_pw_lengths.size() * sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_salt, salt.size()));
    CUDA_CHECK(hipMalloc(&d_ciphertext, ciphertext.size()));
    CUDA_CHECK(hipMalloc(&d_found_flag, sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_found_password, PW_MAXLEN + 1));
    CUDA_CHECK(hipMemcpy(d_pw_data, h_pw_data.data(), h_pw_data.size(), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_pw_lengths, h_pw_lengths.data(), h_pw_lengths.size() * sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_salt, salt.data(), salt.size(), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_ciphertext, ciphertext.data(), ciphertext.size(), hipMemcpyHostToDevice));
    int h_found_flag = 0;
    CUDA_CHECK(hipMemcpy(d_found_flag, &h_found_flag, sizeof(int), hipMemcpyHostToDevice));

    std::cout << "Starting...\n";
    auto start = std::chrono::high_resolution_clock::now();

    int threads_per_block = 64;
    int blocks = (passwords.size() + threads_per_block - 1) / threads_per_block;

    generation_and_validation_kernel<<<blocks, threads_per_block>>>(
        d_pw_data, d_pw_lengths, PW_MAXLEN, d_salt, d_ciphertext, ciphertext.size(),
        11513, 10000, 128, 16,
        d_found_flag, d_found_password, passwords.size(),
        -1
    );
    
    CUDA_CHECK(hipDeviceSynchronize());
    auto stop = std::chrono::high_resolution_clock::now();
    // *** KORRIGIERTE ZEILE 222 ***
    double elapsed_s = std::chrono::duration<double>(stop - start).count();

    CUDA_CHECK(hipMemcpy(&h_found_flag, d_found_flag, sizeof(int), hipMemcpyDeviceToHost));

    std::cout << "\n--------------------------------------------------\n";
    std::cout << "Processing finished in " << std::fixed << std::setprecision(4) << elapsed_s << " seconds.\n";
    if (elapsed_s > 0) std::cout << "Throughput: " << std::fixed << std::setprecision(2) << (passwords.size() / elapsed_s) << " Pw/s.\n";
    
    if (h_found_flag) {
        std::vector<char> h_found_password(PW_MAXLEN + 1, 0);
        CUDA_CHECK(hipMemcpy(h_found_password.data(), d_found_password, PW_MAXLEN + 1, hipMemcpyDeviceToHost));
        std::cout << "\n>>> MATCH FOUND! <<<\n";
        std::cout << "The correct password is: " << std::string(h_found_password.data()) << "\n";
    } else {
        std::cout << "\nNo match found in the list.\n";
    }
    std::cout << "--------------------------------------------------\n";

    hipFree(d_pw_data); hipFree(d_pw_lengths); hipFree(d_salt); hipFree(d_ciphertext);
    hipFree(d_found_flag); hipFree(d_found_password);

    return 0;
}